#include "hip/hip_runtime.h"
/*
 Copyright (c) 2023 The EPIC++ authors

 Permission is hereby granted, free of charge, to any person obtaining a copy of
 this software and associated documentation files (the "Software"), to deal in
 the Software without restriction, including without limitation the rights to
 use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 the Software, and to permit persons to whom the Software is furnished to do so,
 subject to the following conditions:

 The above copyright notice and this permission notice shall be included in all
 copies or substantial portions of the Software.

 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include "ex/MOFF_cu_handler.h"
#include "ex/constants.h"
#include "ex/cu_helpers.cuh"
#include "ex/fft_dx.cuh"
#include "ex/gridder.cuh"
#include "ex/types.hpp"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <nvml.h>
#include <glog/logging.h>

namespace cg = cooperative_groups;
typedef __half2 f16_accum_t;

// Note: For a 40 ms gulp, bfloat takes about a 36.8 ms while half takes about 38 ms.
// However, the loss of precision is larger with bfloat compared to half.
// Hence half is chosen for on-chip accumulation.
// typedef __hip_bfloat162 f16_accum_t;

void
MOFFCuHandler::ResetAntpos(int p_nchan, float* p_antpos_ptr)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_antpos_set) {
        hipFree(m_antpos_cu);
    }
    auto nbytes = LWA_SV_NSTANDS * p_nchan * 3 * sizeof(float);
    hipMalloc(&m_antpos_cu, nbytes);
    hipMemcpy(m_antpos_cu, p_antpos_ptr, nbytes, hipMemcpyHostToDevice);
    is_antpos_set = true;
}

void
MOFFCuHandler::ResetPhases(int p_nchan, float* p_phases_ptr)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_phases_set) {
        hipFree(m_phases_cu);
    }

    auto nbytes = LWA_SV_NSTANDS * p_nchan * LWA_SV_NPOLS * 2 * sizeof(float);
    hipMalloc(&m_phases_cu, nbytes);
    hipMemcpy(m_phases_cu, p_phases_ptr, nbytes, hipMemcpyHostToDevice);

    is_phases_set = true;
}

void
MOFFCuHandler::ResetGcfTex(int p_gcf_tex_dim, float* p_gcf_2D_ptr)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_gcf_tex_set) {
        cuda_check_err(hipFreeArray(m_gcf_tex_arr));
        cuda_check_err(hipDestroyTextureObject(m_gcf_tex));
    }

    hipMallocArray(&m_gcf_tex_arr, &m_gcf_chan_desc, p_gcf_tex_dim, p_gcf_tex_dim);

    memset(&m_gcf_res_desc, 0, sizeof(m_gcf_res_desc));
    m_gcf_res_desc.resType = hipResourceTypeArray;
    m_gcf_res_desc.res.array.array = m_gcf_tex_arr;
    // Specify texture object parameters
    memset(&m_gcf_tex_desc, 0, sizeof(m_gcf_tex_desc));
    m_gcf_tex_desc.addressMode[0] = hipAddressModeClamp;
    m_gcf_tex_desc.addressMode[1] = hipAddressModeClamp;
    // m_gcf_tex_desc.filterMode = hipFilterModePoint;
    m_gcf_tex_desc.filterMode = hipFilterModeLinear;
    m_gcf_tex_desc.readMode = hipReadModeElementType;
    m_gcf_tex_desc.normalizedCoords = 0;

    VLOG(2) << "copying gcf\n";
    const size_t spitch = p_gcf_tex_dim * sizeof(float);
    hipMemcpy2DToArray(m_gcf_tex_arr, 0, 0, p_gcf_2D_ptr, spitch, p_gcf_tex_dim * sizeof(float), p_gcf_tex_dim, hipMemcpyHostToDevice);

    VLOG(2) << "texture set\n";
    hipCreateTextureObject(&m_gcf_tex, &m_gcf_res_desc, &m_gcf_tex_desc, NULL);

    is_gcf_tex_set = true;
    cuda_check_err(hipPeekAtLastError());
}

void
MOFFCuHandler::CreateGulpCuStreams()
{
    cuda_check_err(hipSetDevice(m_device_id));
    m_gulp_custreams.reset();
    m_gulp_custreams = std::make_unique<hipStream_t[]>(m_nstreams);
    for (int i = 0; i < m_nstreams; ++i) {
        hipStreamCreate(m_gulp_custreams.get() + i);
    }
}

void MOFFCuHandler::ResetGcfElem(int p_nchan, int p_support, int p_chan0, float p_delta, int p_grid_size){
    cuda_check_err(hipSetDevice(m_device_id));
    if(is_m_gcf_elem_set){
        cuda_check_err(hipFree(m_gcf_elem));
        is_m_gcf_elem_set = false;
    }
    auto nelements_gcf = (p_support) * (p_support);
    auto nbytes = LWA_SV_NSTANDS * p_nchan * nelements_gcf * sizeof(float);
    cuda_check_err(hipMalloc(&m_gcf_elem, nbytes));
    is_m_gcf_elem_set=true;

    int block_size = (MAX_THREADS_PER_BLOCK/float(nelements_gcf)) ;
    block_size *= nelements_gcf;

    VLOG(2)<<"Pre-computing GCF elements\n"<<p_support<<" "<<block_size<<" "<<nelements_gcf;
    ComputeGcfElements<<<p_nchan, block_size>>>(m_gcf_elem, m_antpos_cu, p_chan0, p_delta, m_gcf_tex,p_grid_size, (p_support), LWA_SV_NSTANDS);
    VLOG(2)<<"Done\n";

    hipDeviceSynchronize();
    cuda_check_err(hipPeekAtLastError());
}

void MOFFCuHandler::GetCorrectionKernel(float* p_out_kernel, int p_support, int p_nchan){
    cuda_check_err(hipSetDevice(m_device_id));
    if(m_nchan_in==0){
        LOG(FATAL)<<"Number of input channels is not set. Unable to compute the averaged kernel\n";
    }
    int nbytes = p_support * p_support * p_nchan * sizeof(float);
    if(is_correction_kernel_set){
        hipFree(m_correction_kernel_d);
        is_correction_kernel_set = false;
    }
    cuda_check_err(hipMalloc(&m_correction_kernel_d, nbytes));
    is_correction_kernel_set = true;

    

    ComputeAvgGriddingKernel<<<p_nchan, LWA_SV_NSTANDS>>>(m_gcf_elem, m_correction_kernel_d ,p_nchan, p_support);

    hipMemcpy(p_out_kernel, m_correction_kernel_d, nbytes, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    cuda_check_err(hipPeekAtLastError());
}
//void MOFFCuHandler::SetCorrectionGrid(float* corr_grid);


void MOFFCuHandler::SetCorrectionGrid(float* p_in_correction_grid, int p_grid_size, int p_nchan){
    cuda_check_err(hipSetDevice(m_device_id));
    int nbytes = p_grid_size * p_grid_size * p_nchan * sizeof(float);
    if(is_correction_grid_set){
        hipFree(m_correction_grid_d);
        is_correction_grid_set=false;
    }
    cuda_check_err(hipMalloc(&m_correction_grid_d, nbytes));
    is_correction_grid_set = true;

    cuda_check_err(hipMemcpy(m_correction_grid_d, p_in_correction_grid, nbytes, hipMemcpyHostToDevice));
    VLOG(2)<<"FINE3\n";
    hipDeviceSynchronize();
    cuda_check_err(hipPeekAtLastError());
}

void
MOFFCuHandler::ResetData(int p_nchan, size_t p_nseq_per_gulp, float* p_antpos_ptr, float* p_phases_ptr)
{
    cuda_check_err(hipSetDevice(m_device_id));
    m_nseq_per_gulp = p_nseq_per_gulp;
    m_nchan_in = p_nchan;

    VLOG(2) << "GPU resetting antpos\n";
    ResetAntpos(p_nchan, p_antpos_ptr);
    cuda_check_err(hipPeekAtLastError());

    VLOG(2) << "GPU resetting phases\n";
    ResetPhases(p_nchan, p_phases_ptr);
    cuda_check_err(hipPeekAtLastError());

    hipDeviceSynchronize();
    cuda_check_err(hipPeekAtLastError());
}

void
MOFFCuHandler::SetImagingKernel()
{   int smemSize;
    hipDeviceGetAttribute(&smemSize, hipDeviceAttributeMaxSharedMemoryPerBlock, m_device_id);
    VLOG(2)<<"Max shared memory per block: "<<smemSize<<" bytes\n";
    cuda_check_err(hipSetDevice(m_device_id));
    // assert(m_out_img_desc.img_size == HALF);
    if (m_out_img_desc.img_size == HALF) {
        VLOG(3)<<"Setting the imaging kernel to 64x64\n";
        VLOG(3)<<"Shared memory size: "<<FFT64x64::shared_memory_size<<" bytes\n";
        VLOG(3)<<FFT64x64::block_dim.x<<" "<<FFT64x64::block_dim.y<<"\n";
        if(use_bf16_accum){
            m_imaging_kernel = GetImagingKernel<FFT64x64, f16_accum_t>(m_support_size);
        }else{
            m_imaging_kernel = GetImagingKernel<FFT64x64, float2>(m_support_size);
        }
        m_img_block_dim = FFT64x64::block_dim;
        m_shared_mem_size = FFT64x64::shared_memory_size*2;
    } else {
         VLOG(3)<<"Setting the imaging kernel to 128x128\n";
        VLOG(3)<<"Shared memory size: "<<FFT128x128::shared_memory_size<<" bytes "<<FFT128x128::elements_per_thread<<"\n";
        VLOG(3)<<FFT64x64::block_dim.x<<" "<<FFT128x128::block_dim.y<<"\n";


        if(use_bf16_accum){
            VLOG(4)<<"ACCUMULATING in 16BIT\n";
            m_imaging_kernel = GetImagingKernel<FFT128x128, f16_accum_t>(m_support_size);
        }else{
            m_imaging_kernel = GetImagingKernel<FFT128x128, float2>(m_support_size);
        }
        m_img_block_dim = FFT128x128::block_dim;
        m_shared_mem_size = FFT128x128::shared_memory_size*1.5;
    }
    hipFuncSetAttribute(reinterpret_cast<const void*>(
          m_imaging_kernel),
          hipFuncAttributeMaxDynamicSharedMemorySize,
          m_shared_mem_size);
}

void
MOFFCuHandler::AllocateFEngGpu(size_t nbytes)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_f_eng_cu_allocated) {
        hipFree(m_f_eng_cu);
        is_f_eng_cu_allocated = false;
    }
    hipMalloc(&m_f_eng_cu, nbytes);
    m_f_eng_bytes = nbytes;
    is_f_eng_cu_allocated = true;
}

void
MOFFCuHandler::AllocateOutImg(size_t p_nbytes)
{
    cuda_check_err(hipSetDevice(m_device_id));
    if (is_out_mem_set) {
        hipFree(m_output_cu);
        is_out_mem_set = false;
    }
    hipMalloc(&m_output_cu, p_nbytes);
    is_out_mem_set = true;
    m_out_img_bytes = p_nbytes;
}

void
MOFFCuHandler::SetImgGridDim()
{
    cuda_check_err(hipSetDevice(m_device_id));
    assert((void("Number of channels per stream cannot be zero"), m_nchan_per_stream > 0));
    if (m_nchan_per_stream > 0) {
        m_img_grid_dim = dim3(m_nchan_per_stream, 1, 1);
    }
}

void
MOFFCuHandler::ProcessGulp(uint8_t* p_data_ptr, float* p_out_ptr, bool p_first, bool p_last, int p_chan0, float p_delta)
{
    cuda_check_err(hipSetDevice(m_device_id));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    VLOG(2)<<"FEng bytes per stream: "<<m_nbytes_f_eng_per_stream<<". OutImg bytes per stream: "<<m_nbytes_out_img_per_stream<<". Chan per stream: "<<m_nchan_per_stream<<". NStreams: "<<m_nstreams<<"\n";
    VLOG(2)<<"Nseq per gulp: "<<m_nseq_per_gulp<<"\n";
    for (int i = 0; i < m_nstreams; ++i) {
        int f_eng_dat_offset = i * m_nbytes_f_eng_per_stream;
        int output_img_offset = i * m_nbytes_out_img_per_stream/sizeof(float);
        auto stream_i = *(m_gulp_custreams.get() + i);
        int chan_offset = i * m_nchan_per_stream;

        void* args[] = {
            &m_f_eng_cu, &m_antpos_cu, &m_phases_cu, &m_nseq_per_gulp, &m_nchan_in, &m_gcf_tex, &m_output_cu, &chan_offset, &p_first, &m_gcf_elem, &m_correction_grid_d
        };

        cuda_check_err(
          hipMemcpyAsync(
            (void*)(m_f_eng_cu + f_eng_dat_offset),
            (void*)(p_data_ptr + f_eng_dat_offset),
            m_nbytes_f_eng_per_stream,
            hipMemcpyHostToDevice,
            stream_i));
        VLOG(2)<<"Launching the kernel\n";
        if(m_imaging_kernel==nullptr){
            VLOG(2)<<"Null imaging kernel\n";
        }
        VLOG(2)<<m_img_grid_dim.x<<" "<<m_img_grid_dim.y<<" "<<m_img_block_dim.x<<" "<<m_img_block_dim.y<<" "<<m_shared_mem_size<<std::endl;
        cuda_check_err(hipLaunchKernel(reinterpret_cast<const void*>(m_imaging_kernel), m_img_grid_dim, m_img_block_dim, args, m_shared_mem_size, stream_i));

        VLOG(2)<<"chan0: "<<p_chan0<<" delta: "<<p_delta<<"\n";

        VLOG(2)<<i<<" "<<output_img_offset<<" "<<"\n";
        if (p_last) {
            cuda_check_err(hipMemcpyAsync((void*)(p_out_ptr + output_img_offset), (void*)(m_output_cu + output_img_offset), m_nbytes_out_img_per_stream, hipMemcpyDeviceToHost, stream_i));
        }
    }

    if (p_last) {
        VLOG(2)<<"Syncing the kernels\n";
        for (int i = 0; i < m_nstreams; ++i) {
            cuda_check_err(hipStreamSynchronize(*(m_gulp_custreams.get() + i)));
        }
        VLOG(2)<<"Syncing done\n";
        cuda_check_err(hipPeekAtLastError());
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    PrometheusExporter::ObserveRunTimeValue(m_gulp_exec_gauge_id, milliseconds);
    VLOG(1) << "Gulp processing time (ms) on GPU("<<m_device_id<<"): " << milliseconds << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

}

void
MOFFCuHandler::DestroyTextures(hipArray_t& p_tex_arr, hipTextureObject_t& p_tex_obj)
{
    cuda_check_err(hipSetDevice(m_device_id));
    hipFreeArray(p_tex_arr);
    hipDestroyTextureObject(p_tex_obj);
}

MOFFCuHandler::~MOFFCuHandler()
{
    cuda_check_err(hipSetDevice(m_device_id));
    // DestroyTextures(m_antpos_tex_arr, m_antpos_tex);
    // DestroyTextures(m_phases_tex_arr, m_phases_tex);
    if (is_antpos_set) {
        hipFree(m_antpos_cu);
    }
    if (is_phases_set) {
        hipFree(m_phases_cu);
    }

    if(is_correction_kernel_set){
        hipFree(m_correction_kernel_d);
    }

    if(is_correction_grid_set){
        hipFree(m_correction_grid_d);
    }

    if(is_m_gcf_elem_set){
        hipFree(m_gcf_elem);
    }

    if(is_out_mem_set){
        hipFree(m_output_cu);
    }

    if(is_f_eng_cu_allocated){
        hipFree(m_f_eng_cu);
    }
}
